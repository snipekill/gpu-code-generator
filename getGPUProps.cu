
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
int main(int argc, char *argv[])
{
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    cout<< "SHMEM_PER_BLOCK = " << prop.sharedMemPerBlock<<"\n";
    cout<< "WARP_SIZE = " << prop.warpSize<<"\n";
    cout<< "MAX_THREADS_PER_BLOCK = " << prop.maxThreadsPerBlock<<"\n"; 

  }
  
  return 0;
}